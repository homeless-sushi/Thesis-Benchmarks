#include "hip/hip_runtime.h"
#include "HMLSS_BFS/BFS.h"
#include "HMLSS_BFS/BFSKnobs.h"
#include "HMLSS_BFS/BFS_CUDA.h"

#include <vector>

#include <hip/hip_runtime.h>

#include "HMLSS_Graph/Graph.h"

namespace BFS
{
    BFSCUDA::BFSCUDA(Graph::Graph& graph, unsigned int source) :
        BFSResult(graph, source)
    {
        hipMalloc(&edgeOffsetsDevice_, sizeof(unsigned int)*graph.edgeOffsets.capacity());
        hipMemcpy(edgeOffsetsDevice_, graph.edgeOffsets.data(), sizeof(unsigned int)*graph.edgeOffsets.capacity(), hipMemcpyKind::hipMemcpyHostToDevice);

        hipMalloc(&edgesDevice_, sizeof(unsigned int)*graph.edges.capacity());
        hipMemcpy(edgesDevice_, graph.edges.data(), sizeof(unsigned int)*graph.edges.capacity(), hipMemcpyKind::hipMemcpyHostToDevice);

        hipMalloc(&costsDevice_, sizeof(int)*graph.nVertices);
        hipMemset(costsDevice_, -1, sizeof(unsigned int)*graph.nVertices);
        hipMemset(costsDevice_ + source, 0, sizeof(unsigned int));

        hipMalloc(&doneDevice_, sizeof(bool));
        hipMemset(doneDevice_, true, sizeof(bool));
    }

    BFSCUDA::BFSCUDA(BFSResult bfsCPU) :
        BFSCUDA(bfsCPU.graph, bfsCPU.source)
    {}

    BFSCUDA::~BFSCUDA() 
    {
        hipFree(edgeOffsetsDevice_);
        hipFree(edgesDevice_);
        hipFree(costsDevice_);
        hipFree(doneDevice_);
    }

    namespace
    {
        __global__
        void CUDAkernel(
            unsigned int nVertices,
            unsigned int chunkSize,
            unsigned int *edgeOffsets,
            unsigned int *edges,
            int *costs,
            int currCost,
            bool *done)
        {
            const unsigned int startNode = blockIdx.x*chunkSize;
            const unsigned int stopNode = min(startNode + chunkSize, nVertices);
            for(unsigned int fromNode = startNode; fromNode < stopNode; fromNode++) {
                if(costs[fromNode] == currCost) {
                    const unsigned int nodeEdgesStart = edgeOffsets[fromNode];
                    const unsigned int nodeEdgesEnd = edgeOffsets[fromNode+1];
                    for(unsigned int i = nodeEdgesStart + threadIdx.x; i < nodeEdgesEnd; i+=blockDim.x) {
                        const unsigned int toNode = edges[i];
                        if(costs[toNode] == -1) {
                            costs[toNode] = currCost + 1;
                            *done = false;
                        }
                    }
                }
            }
        }
    }

    bool BFSCUDA::kernel(BFSKnobs::Knobs knobs)
    {
        hipMemset(doneDevice_, true, sizeof(bool));

        const unsigned int blockDim = knobs.gpuKnobs.blockSize;
        const unsigned int chunkSize = blockDim * knobs.gpuKnobs.chunkFactor;
        const unsigned int gridDim = (graph.nVertices + blockDim - 1)/chunkSize + 1;
        CUDAkernel<<<gridDim, blockDim>>>(graph.nVertices, chunkSize, edgeOffsetsDevice_, edgesDevice_, costsDevice_, currentCost, doneDevice_);
        currentCost++;

        bool done;
        hipMemcpy(&done, doneDevice_, sizeof(bool), hipMemcpyKind::hipMemcpyDeviceToHost);
        return done;
    }

    const std::vector<int>& BFSCUDA::costs() 
    {
        costsHost_.reserve(graph.nVertices);
        costsHost_.resize(costsHost_.capacity());
        hipMemcpy(costsHost_.data(), costsDevice_, sizeof(int)*graph.nVertices, hipMemcpyKind::hipMemcpyDeviceToHost);
        return costsHost_;
    };
}