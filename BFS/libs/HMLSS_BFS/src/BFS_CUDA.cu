#include "hip/hip_runtime.h"
#include "HMLSS_BFS/BFS.h"
#include "HMLSS_BFS/BFSKnobs.h"
#include "HMLSS_BFS/BFS_CUDA.h"

#include <vector>

#include <hip/hip_runtime.h>

#include "HMLSS_Graph/Graph.h"

namespace BFS
{
    BFSCUDA::BFSCUDA(Graph::Graph& graph, unsigned int source) :
        BFSResult(graph, source)
    {
        hipMalloc(&edgeOffsetsDevice_, sizeof(unsigned int)*graph.edgeOffsets.size());
        hipMemcpy(edgeOffsetsDevice_, graph.edgeOffsets.data(), sizeof(unsigned int)*graph.edgeOffsets.size(), hipMemcpyKind::hipMemcpyHostToDevice);

        hipMalloc(&edgesDevice_, sizeof(unsigned int)*graph.edges.size());
        hipMemcpy(edgesDevice_, graph.edges.data(), sizeof(unsigned int)*graph.edges.size(), hipMemcpyKind::hipMemcpyHostToDevice);

        memset(&edgeOffsetsTexture_, 0, sizeof(hipTextureObject_t));
        memset(&edgesTexture_, 0, sizeof(hipTextureObject_t));

        hipMalloc(&costsDevice_, sizeof(int)*graph.nVertices);
        hipMemset(costsDevice_, -1, sizeof(unsigned int)*graph.nVertices);
        hipMemset(costsDevice_ + source, 0, sizeof(unsigned int));

        hipMalloc(&doneDevice_, sizeof(bool));
        hipMemset(doneDevice_, true, sizeof(bool));
    }

    BFSCUDA::BFSCUDA(
        Graph::Graph& graph,
        unsigned int source,
        int currentCost,
        std::vector<int> costs) :
        BFSResult(graph, source)
    {
        this->currentCost = currentCost;

        hipMalloc(&edgeOffsetsDevice_, sizeof(unsigned int)*graph.edgeOffsets.size());
        hipMemcpy(edgeOffsetsDevice_, graph.edgeOffsets.data(), sizeof(unsigned int)*graph.edgeOffsets.size(), hipMemcpyKind::hipMemcpyHostToDevice);

        hipMalloc(&edgesDevice_, sizeof(unsigned int)*graph.edges.size());
        hipMemcpy(edgesDevice_, graph.edges.data(), sizeof(unsigned int)*graph.edges.size(), hipMemcpyKind::hipMemcpyHostToDevice);

        memset(&edgeOffsetsTexture_, 0, sizeof(hipTextureObject_t));
        memset(&edgesTexture_, 0, sizeof(hipTextureObject_t));

        hipMalloc(&costsDevice_, sizeof(int)*costs.size());
        hipMemcpy(costsDevice_, costs.data(), sizeof(int)*costs.size(), hipMemcpyKind::hipMemcpyHostToDevice);

        hipMalloc(&doneDevice_, sizeof(bool));
        hipMemset(doneDevice_, true, sizeof(bool));
    }

    BFSCUDA::BFSCUDA(BFSResult bfsCPU) :
        BFSCUDA(bfsCPU.graph, bfsCPU.source, bfsCPU.currentCost, bfsCPU.costs())
    {}

    BFSCUDA::~BFSCUDA() 
    {
        hipFree(edgeOffsetsDevice_);
        hipFree(edgesDevice_);
        hipDestroyTextureObject(edgeOffsetsTexture_);
        hipDestroyTextureObject(edgesTexture_);
        hipFree(costsDevice_);
        hipFree(doneDevice_);
    }

    namespace
    {
        void createTextureObject(unsigned int* src, hipTextureObject_t* dst, size_t size)
        {
            hipResourceDesc resourceDesc;
            memset(&resourceDesc, 0, sizeof(resourceDesc));
            resourceDesc.resType = hipResourceTypeLinear;
            resourceDesc.res.linear.devPtr = src;
            resourceDesc.res.linear.desc.f = hipChannelFormatKindUnsigned;
            resourceDesc.res.linear.desc.x = sizeof(unsigned int)*CHAR_BIT;
            resourceDesc.res.linear.sizeInBytes = sizeof(unsigned int)*size;
            hipTextureDesc texDesc;
            memset(&texDesc, 0, sizeof(texDesc));
            texDesc.readMode = hipReadModeElementType;
            hipCreateTextureObject(dst, &resourceDesc, &texDesc, NULL);
        }

        __global__
        void CUDAkernel(
            unsigned int nVertices,
            unsigned int chunkSize,
            unsigned int *edgeOffsetsDevice,
            unsigned int *edgesDevice,
            hipTextureObject_t edgeOffsetsTexture,
            hipTextureObject_t edgesTexture,
            BFSKnobs::GPUKnobs::MEMORY_TYPE edgeOffsetsType,
            BFSKnobs::GPUKnobs::MEMORY_TYPE edgesType,
            int *costs,
            int currCost,
            bool *done)
        {
            const unsigned int startNode = blockIdx.x*chunkSize;
            const unsigned int stopNode = min(startNode + chunkSize, nVertices);
            for(unsigned int fromNode = startNode; fromNode < stopNode; fromNode++) {
                if(costs[fromNode] == currCost) {
                    unsigned int nodeEdgesStart;
                    unsigned int nodeEdgesEnd;
                    switch (edgeOffsetsType)
                    {
                        case BFSKnobs::GPUKnobs::MEMORY_TYPE::TEXTURE_MEMORY:
                            nodeEdgesStart = tex1Dfetch<unsigned int>(edgeOffsetsTexture, fromNode);
                            nodeEdgesEnd = tex1Dfetch<unsigned int>(edgeOffsetsTexture, fromNode+1);
                            break;
                        
                        default:
                            nodeEdgesStart = edgeOffsetsDevice[fromNode];
                            nodeEdgesEnd = edgeOffsetsDevice[fromNode+1];
                            break;
                    }
                    for(unsigned int i = nodeEdgesStart + threadIdx.x; i < nodeEdgesEnd; i+=blockDim.x) {
                        unsigned int toNode;
                        switch (edgesType)
                        {
                            case BFSKnobs::GPUKnobs::MEMORY_TYPE::TEXTURE_MEMORY:
                                toNode = tex1Dfetch<unsigned int>(edgesTexture, i);
                                break;
                            
                            default:
                                toNode = edgesDevice[i];
                                break;
                        }
                        if(costs[toNode] == -1) {
                            costs[toNode] = currCost + 1;
                            *done = false;
                        }
                    }
                }
            }
        }
    }

    bool BFSCUDA::kernel(BFSKnobs::Knobs knobs)
    {
        hipMemset(doneDevice_, true, sizeof(bool));

        if(knobs.gpuKnobs.edgeOffsets == BFSKnobs::GPUKnobs::MEMORY_TYPE::TEXTURE_MEMORY && edgeOffsetsTexture_ == 0)
            createTextureObject(edgeOffsetsDevice_, &edgeOffsetsTexture_, graph.edgeOffsets.size());
        if(knobs.gpuKnobs.edges == BFSKnobs::GPUKnobs::MEMORY_TYPE::TEXTURE_MEMORY && edgesTexture_ == 0)
            createTextureObject(edgesDevice_, &edgesTexture_, graph.edges.size());

        const unsigned int blockDim = knobs.gpuKnobs.blockSize;
        const unsigned int chunkSize = blockDim * knobs.gpuKnobs.chunkFactor;
        const unsigned int gridDim = (graph.nVertices + blockDim - 1)/chunkSize + 1;

        CUDAkernel<<<gridDim, blockDim>>>(
            graph.nVertices, 
            chunkSize,
            edgeOffsetsDevice_, 
            edgesDevice_,
            edgeOffsetsTexture_,
            edgesTexture_,
            knobs.gpuKnobs.edgeOffsets,
            knobs.gpuKnobs.edges,
            costsDevice_,
            currentCost,
            doneDevice_);
        
        currentCost++;
        bool done;
        hipMemcpy(&done, doneDevice_, sizeof(bool), hipMemcpyKind::hipMemcpyDeviceToHost);
        return done;
    }

    const std::vector<int>& BFSCUDA::costs() 
    {
        costsHost_.reserve(graph.nVertices);
        costsHost_.resize(costsHost_.capacity());
        hipMemcpy(costsHost_.data(), costsDevice_, sizeof(int)*graph.nVertices, hipMemcpyKind::hipMemcpyDeviceToHost);
        return costsHost_;
    };
}