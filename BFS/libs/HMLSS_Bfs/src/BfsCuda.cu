#include "hip/hip_runtime.h"
#include "HMLSS_Bfs/Bfs.h"
#include "HMLSS_Bfs/BfsKnobs.h"
#include "HMLSS_Bfs/BfsCuda.h"

#include <vector>

#include <hip/hip_runtime.h>

#include "HMLSS_Graph/Graph.h"

namespace BFS
{
    BfsCuda::BfsCuda(GpuKnobs knobs, Graph::Graph& graph, unsigned int source) :
        BfsResult(graph, source),
        knobs_(knobs)
    {
        hipMalloc(&edgeOffsetsDevice_, sizeof(unsigned int)*graph.edgeOffsets.size());
        hipMemcpy(edgeOffsetsDevice_, graph.edgeOffsets.data(), sizeof(unsigned int)*graph.edgeOffsets.size(), hipMemcpyKind::hipMemcpyHostToDevice);

        hipMalloc(&edgesDevice_, sizeof(unsigned int)*graph.edges.size());
        hipMemcpy(edgesDevice_, graph.edges.data(), sizeof(unsigned int)*graph.edges.size(), hipMemcpyKind::hipMemcpyHostToDevice);

        memset(&edgeOffsetsTexture_, 0, sizeof(hipTextureObject_t));
        memset(&edgesTexture_, 0, sizeof(hipTextureObject_t));

        hipMalloc(&costsDevice_, sizeof(int)*graph.nVertices);
        hipMemset(costsDevice_, -1, sizeof(unsigned int)*graph.nVertices);
        hipMemset(costsDevice_ + source, 0, sizeof(unsigned int));

        hipMalloc(&doneDevice_, sizeof(bool));
        hipMemset(doneDevice_, true, sizeof(bool));
    }

    BfsCuda::~BfsCuda() 
    {
        hipFree(edgeOffsetsDevice_);
        hipFree(edgesDevice_);
        hipDestroyTextureObject(edgeOffsetsTexture_);
        hipDestroyTextureObject(edgesTexture_);
        hipFree(costsDevice_);
        hipFree(doneDevice_);
    }

    namespace
    {
        void createTextureObject(unsigned int* src, hipTextureObject_t* dst, size_t size)
        {
            hipResourceDesc resourceDesc;
            memset(&resourceDesc, 0, sizeof(resourceDesc));
            resourceDesc.resType = hipResourceTypeLinear;
            resourceDesc.res.linear.devPtr = src;
            resourceDesc.res.linear.desc.f = hipChannelFormatKindUnsigned;
            resourceDesc.res.linear.desc.x = sizeof(unsigned int)*CHAR_BIT;
            resourceDesc.res.linear.sizeInBytes = sizeof(unsigned int)*size;
            hipTextureDesc texDesc;
            memset(&texDesc, 0, sizeof(texDesc));
            texDesc.readMode = hipReadModeElementType;
            hipCreateTextureObject(dst, &resourceDesc, &texDesc, NULL);
        }

        __global__
        void CUDAkernel(
            unsigned int nVertices,
            unsigned int chunkSize,
            unsigned int *edgeOffsetsDevice,
            unsigned int *edgesDevice,
            hipTextureObject_t edgeOffsetsTexture,
            hipTextureObject_t edgesTexture,
            BFS::GpuKnobs::MEMORY_TYPE edgeOffsetsType,
            BFS::GpuKnobs::MEMORY_TYPE edgesType,
            int *costs,
            int currCost,
            bool *done)
        {
            const unsigned int startNode = blockIdx.x*chunkSize;
            const unsigned int stopNode = min(startNode + chunkSize, nVertices);
            for(unsigned int fromNode = startNode; fromNode < stopNode; fromNode++) {
                if(costs[fromNode] == currCost) {
                    unsigned int nodeEdgesStart;
                    unsigned int nodeEdgesEnd;
                    switch (edgeOffsetsType)
                    {
                        case BFS::GpuKnobs::MEMORY_TYPE::TEXTURE_MEMORY:
                            nodeEdgesStart = tex1Dfetch<unsigned int>(edgeOffsetsTexture, fromNode);
                            nodeEdgesEnd = tex1Dfetch<unsigned int>(edgeOffsetsTexture, fromNode+1);
                            break;
                        
                        default:
                            nodeEdgesStart = edgeOffsetsDevice[fromNode];
                            nodeEdgesEnd = edgeOffsetsDevice[fromNode+1];
                            break;
                    }
                    for(unsigned int i = nodeEdgesStart + threadIdx.x; i < nodeEdgesEnd; i+=blockDim.x) {
                        unsigned int toNode;
                        switch (edgesType)
                        {
                            case BFS::GpuKnobs::MEMORY_TYPE::TEXTURE_MEMORY:
                                toNode = tex1Dfetch<unsigned int>(edgesTexture, i);
                                break;
                            
                            default:
                                toNode = edgesDevice[i];
                                break;
                        }
                        if(costs[toNode] == -1) {
                            costs[toNode] = currCost + 1;
                            *done = false;
                        }
                    }
                }
            }
        }
    }

    bool BfsCuda::kernel()
    {
        hipMemset(doneDevice_, true, sizeof(bool));

        if(knobs_.edgeOffsets == BFS::GpuKnobs::MEMORY_TYPE::TEXTURE_MEMORY && edgeOffsetsTexture_ == 0)
            createTextureObject(edgeOffsetsDevice_, &edgeOffsetsTexture_, graph.edgeOffsets.size());
        if(knobs_.edges == BFS::GpuKnobs::MEMORY_TYPE::TEXTURE_MEMORY && edgesTexture_ == 0)
            createTextureObject(edgesDevice_, &edgesTexture_, graph.edges.size());

        const unsigned int blockDim = knobs_.blockSize;
        const unsigned int chunkSize = blockDim * knobs_.chunkFactor;
        const unsigned int gridDim = (graph.nVertices + blockDim - 1)/chunkSize + 1;

        CUDAkernel<<<gridDim, blockDim>>>(
            graph.nVertices, 
            chunkSize,
            edgeOffsetsDevice_, 
            edgesDevice_,
            edgeOffsetsTexture_,
            edgesTexture_,
            knobs_.edgeOffsets,
            knobs_.edges,
            costsDevice_,
            currentCost,
            doneDevice_);
        
        currentCost++;
        bool done;
        hipMemcpy(&done, doneDevice_, sizeof(bool), hipMemcpyKind::hipMemcpyDeviceToHost);
        return done;
    }

    const std::vector<int>& BfsCuda::costs() 
    {
        costsHost_.reserve(graph.nVertices);
        costsHost_.resize(costsHost_.capacity());
        hipMemcpy(costsHost_.data(), costsDevice_, sizeof(int)*graph.nVertices, hipMemcpyKind::hipMemcpyDeviceToHost);
        return costsHost_;
    };
}